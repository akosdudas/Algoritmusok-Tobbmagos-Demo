#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <Windows.h>

__global__ void matrixMultiplyKernel(float* result, const float* a, const float* b, const int row, const int col)
{
	int i = blockIdx.x;
    int j = threadIdx.x;

	float temp = 0;
	for (int k = 0; k < col; ++k)
		temp += a[i*col+k] * b[k*row+j];
	result[i*col+j] = temp;
}

float* createTestData( int row, int col )
{
	float* data = new float[row*col];
	for (int i = 0; i < row; ++i)
		for (int j = 0; j < col; ++j)
			data[i*col+j] = (float)(i+1) / (j+1);
	return data;
}

void main()
{
	const int row = 1024;
	const int col = 1024;

	float* a = createTestData(row, col);
	float* b = createTestData(col, row);

	float* result = new float[row*row];

	unsigned __int64 start1 = GetTickCount64();

    float* dev_a = 0;
    float* dev_b = 0;
    float* dev_result = 0;

    hipMalloc((void**)&dev_a, row*col*sizeof(float));
    hipMalloc((void**)&dev_b, col*row*sizeof(float));
    hipMalloc((void**)&dev_result, row*row*sizeof(float));
    hipMemcpy(dev_a, a, row*col*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, col*row*sizeof(float), hipMemcpyHostToDevice);

	unsigned __int64 start2 = GetTickCount64();

    matrixMultiplyKernel<<<row,row>>>(dev_result, dev_a, dev_b, row, col);

    hipDeviceSynchronize();

	printf( "%d\n", GetTickCount64() - start2 );

	hipMemcpy(result, dev_result, row*row*sizeof(float), hipMemcpyDeviceToHost);

	delete[] result;
    hipFree(dev_result);
    hipFree(dev_a);
    hipFree(dev_b);
    
	printf( "%d\n", GetTickCount64() - start1 );

	hipDeviceReset();

	delete[] a;
	delete[] b;
}
